


#include<stdio.h>
//#include<cuda.h>
#include<hip/hip_runtime.h>


#define N 32
#define T 32 // max threads per block
#include <stdio.h>
__global__ void vecAdd (int *a, int *b, int *c);
int main() {
int a[N], b[N], c[N];
int *dev_a, *dev_b, *dev_c;
// initialize a and b with real values (NOT SHOWN)
int size = N * sizeof(int);
hipMalloc((void**)&dev_a, size);
hipMalloc((void**)&dev_b, size);
hipMalloc((void**)&dev_c, size);
hipMemcpy(dev_a, a, size,hipMemcpyHostToDevice);
hipMemcpy(dev_b, b, size,hipMemcpyHostToDevice);
vecAdd<<<(int)ceil(N/T),T>>>(dev_a,dev_b,dev_c);
hipMemcpy(c, dev_c, size,hipMemcpyDeviceToHost);
hipFree(dev_a);
hipFree(dev_b);
hipFree(dev_c);
exit (0);
}
__global__ void vecAdd (int *a, int *b, int *c) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < N) {
c[i] = a[i] + b[i];
printf("%d",c[i]);
}
}
